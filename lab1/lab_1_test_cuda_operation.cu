
#include <hip/hip_runtime.h>
#include<stdio.h>

#define cudaCheckError() {                                                       \
    hipError_t e=hipGetLastError();                                            \
    if(e!=hipSuccess) {                                                         \
        printf("CUDA error %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(e)); \
        exit(1);                                                                 \
    }                                                                            \
}

__global__ void kernel(double *arr1, double *arr2, double *ans, int n){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        ans[idx] = (arr1[idx] < arr2[idx]) ? arr1[idx] : arr2[idx];
    }
}

void readVector(double* arr, int n){
    for (int i = 0; i < n; i++) {
        scanf("%lf", &arr[i]);
    }
}

void printVector(double* arr, int n){
    for (int i = 0; i < n; i++) {
        printf("%f ", arr[i]);
    }
    printf("\n");
}

int main(){
    int n;
    scanf("%d", &n);
    
    double *arr1 = (double*)malloc(sizeof(double) * n);
    double *arr2 = (double*)malloc(sizeof(double) * n);
    double *ans =  (double*)malloc(sizeof(double) * n);
    readVector(arr1, n);
    readVector(arr2, n);
    
    double *dev_arr1, *dev_arr2, *dev_ans;
    hipMalloc(&dev_arr1, sizeof(double) * n); cudaCheckError();
    hipMalloc(&dev_arr2, sizeof(double) * n); cudaCheckError();
    hipMalloc(&dev_ans, sizeof(double) * n); cudaCheckError();
    
    hipMemcpy(dev_arr1, arr1, sizeof(double) * n, hipMemcpyHostToDevice); cudaCheckError();
    hipMemcpy(dev_arr2, arr2, sizeof(double) * n, hipMemcpyHostToDevice); cudaCheckError();
    
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    kernel<<<blocksPerGrid, threadsPerBlock>>>(dev_arr1, dev_arr2, dev_ans, n);
    cudaCheckError();
    
    hipDeviceSynchronize(); cudaCheckError();
    
    hipMemcpy(ans, dev_ans, sizeof(double) * n, hipMemcpyDeviceToHost); cudaCheckError();

    printVector(ans,n);

    free(arr1);
    free(arr2);
    free(ans);
    hipFree(dev_arr1);
    hipFree(dev_arr2);
    hipFree(dev_ans);

    return 0;
}
