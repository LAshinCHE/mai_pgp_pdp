
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void kernel(double *arr1, double *arr2, int n){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = blockDim.x * gridDim.x;
    while (idx < n) {
        arr1[idx] = (arr1[idx] < arr2[idx]) ? arr1[idx] : arr2[idx];
        idx += offset;
    }
}

#define BENCHMARK

const int BLOCKS = 32;
const int THREADS = 32;

void readVector(double* arr, int n){
    for (int i = 0; i < n; i++) {
        scanf("%lf", &arr[i]);
    }
}

void printVector(double* arr, int n){
    for (int i = 0; i < n; i++) {
        printf("%.10lf ", arr[i]);
    }
    printf("\n");
}

int main(){
    int n;
    scanf("%d", &n);

    double *arr1 = (double*)malloc(sizeof(double) * n);
    double *arr2 = (double*)malloc(sizeof(double) * n);
    readVector(arr1, n);
    readVector(arr2, n);

    double *dev_arr1, *dev_arr2;
    hipMalloc(&dev_arr1, sizeof(double) * n);
    hipMalloc(&dev_arr2, sizeof(double) * n);

    hipMemcpy(dev_arr1, arr1, sizeof(double) * n, hipMemcpyHostToDevice);
    hipMemcpy(dev_arr2, arr2, sizeof(double) * n, hipMemcpyHostToDevice);

    #ifdef BENCHMARK
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    #endif /* BENCHMARK */


    kernel<<<BLOCKS, THREADS>>>(dev_arr1, dev_arr2, n);

    hipDeviceSynchronize();

    #ifdef BENCHMARK
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float time;
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("time = %f ms\n", time);
    #endif /* BENCHMARK */


    hipMemcpy(arr1, dev_arr1, sizeof(double) * n, hipMemcpyDeviceToHost);
    #ifndef BENCHMARK
    printVector(arr1,n);
    #endif

    free(arr1);
    free(arr2);
    hipFree(dev_arr1);
    hipFree(dev_arr2);

    return 0;
}