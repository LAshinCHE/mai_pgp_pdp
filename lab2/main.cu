
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <iostream>

#define CSC(call)  									                \
do {											                    \
	hipError_t res = call;							                \
	if (res != hipSuccess) {							            \
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
				__FILE__, __LINE__, hipGetErrorString(res));		\
		exit(0);								                    \
	}										                        \
} while(0)


__global__ void kernel(hipTextureObject_t tex, uchar4 *out, int width, int height) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;
    int offsetx = blockDim.x * gridDim.x;
    int offsety = blockDim.y * gridDim.y;
    int x, y;
    uchar4 p;
    for(y = idy; y < height; y += offsety){
        for(x = idx; x < width; x += offsetx) {
            double w[3][3];

            for (int i = 0; i < 3; ++i) {
                for (int j = 0; j < 3; ++j) {
                    int xi = x + i - 1;
                    int yj = y + j - 1;
                    uchar4 p = tex2D<uchar4>(tex, xi, yj);
                    w[i][j] = 0.299 * p.x + 0.587 * p.y + 0.114 * p.z;
                }
            }

            double gx = w[0][2] + 2 * w[1][2] + w[2][2] - w[0][0] - 2 * w[1][0] - w[2][0];
            double gy = w[2][0] + 2 * w[2][ 1] + w[2][2] - w[0][0] - 2 * w[0][1] - w[0][2];
            int gf = min(255, int(sqrt(gx*gx + gy*gy)));

            out[y *width + x] = make_uchar4(gf, gf, gf, gf);
        }
    }
}




int main() {
    int w, h;
    std::string input;
    std::cin >> input;
    std::string output;
    std::cin >> output;
    FILE *fp = fopen(input.c_str(), "rb");
 	fread(&w, sizeof(int), 1, fp);
	fread(&h, sizeof(int), 1, fp);
 	uchar4 *data = (uchar4 *)malloc(sizeof(uchar4) * w * h);
    fread(data, sizeof(uchar4), w * h, fp);
    fclose(fp);

    hipArray *arr;
    hipChannelFormatDesc ch = hipCreateChannelDesc<uchar4>();
    CSC(hipMallocArray(&arr, &ch, w, h));
    CSC(hipMemcpy2DToArray(arr, 0, 0, data, w * sizeof(uchar4), w * sizeof(uchar4), h, hipMemcpyHostToDevice));

    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = arr;

    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = false;

    hipTextureObject_t tex = 0;
    CSC(hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL));

    uchar4 *dev_out;
    CSC(hipMalloc(&dev_out, sizeof(uchar4) * w * h));

    kernel<<< dim3(16, 16), dim3(32, 32) >>>(tex, dev_out, w, h);
    CSC(hipDeviceSynchronize());
    CSC(hipGetLastError());

    CSC(hipMemcpy(data, dev_out, sizeof(uchar4) * w * h, hipMemcpyDeviceToHost));

    CSC(hipDestroyTextureObject(tex));
    CSC(hipFreeArray(arr));
    CSC(hipFree(dev_out));

    fp = fopen(output.c_str(), "wb");
	fwrite(&w, sizeof(int), 1, fp);
	fwrite(&h, sizeof(int), 1, fp);
	fwrite(data, sizeof(uchar4), w * h, fp);
	fclose(fp);

    free(data);
    return 0;
}